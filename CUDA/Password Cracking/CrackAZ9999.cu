#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
/****************************************************************************
  This program gives an example of a poor way to implement a password cracker
  in CUDA C. It is poor because it acheives this with just one thread, which
  is obviously not good given the scale of parallelism available to CUDA
  programs.
  
  The intentions of this program are:
    1) Demonstrate the use of __device__ and __global__ functions
    2) Enable a simulation of password cracking in the absence of library 
       with equivalent functionality to libcrypt. The password to be found
       is hardcoded into a function called is_a_match.   

  Compile and run with:
    nvcc -o CrackAZ9999 CrackAZ9999.cu
    ./CrackAZ9999 
   
  Dr Kevan Buckley, University of Wolverhampton, 2018
*****************************************************************************/

/****************************************************************************
  This function returns 1 if the attempt at cracking the password is 
  identical to the plain text password string stored in the program. 
  Otherwise,it returns 0.
*****************************************************************************/

__device__ int is_a_match(char *attempt) {
	char plain_password1[] = "PR1234";
	char plain_password2[] = "AJ5478";
	char plain_password3[] = "WO9012";
	char plain_password4[] = "SE3456";


	char *a = attempt;
	char *b = attempt;
	char *c = attempt;
	char *d = attempt;
	char *pw1 = plain_password1;
	char *pw2 = plain_password2;
	char *pw3 = plain_password3;
	char *pw4 = plain_password4;

	while(*a == *pw1) { 
		if(*a == '\0') 
		{
			printf("Password: %s\n",plain_password1);
			break;
		}

		a++;
		pw1++;
	}
	
	while(*b == *pw2) { 
		if(*b == '\0') 
		{
			printf("Password: %s\n",plain_password2);
			break;
		}

		b++;
		pw2++;
	}

	while(*c == *pw3) { 
		if(*c == '\0') 
		{
			printf("Password: %s\n",plain_password3);
			break;
		}

		c++;
		pw3++;
	}

	while(*d == *pw4) { 
		if(*d == '\0') 
		{
			printf("Password: %s\n",plain_password4);
			return 1;
		}

		d++;
		pw4++;
	}
	return 0;

}

__global__ void  kernel() {
	char i1,i2,i3,i4;

	char password[7];
	password[6] = '\0';

	int i = blockIdx.x+65;
	int j = threadIdx.x+65;
	char firstMatch = i; 
	char secondMatch = j; 

	password[0] = firstMatch;
	password[1] = secondMatch;
	for(i1='0'; i1<='9'; i1++){
		for(i2='0'; i2<='9'; i2++){
			for(i3='0'; i3<='9'; i3++){
				for(i4='0'; i4<='9'; i4++){
					password[2] = i1;
					password[3] = i2;
					password[4] = i3;
					password[5] = i4; 
					if(is_a_match(password)) {
					} 
					else {
	     			//printf("tried: %s\n", password);		  
					}
				}
			}
		}
	}
}

// Calculating the time
int time_difference(struct timespec *start, 
                    struct timespec *finish, 
                    long long int *difference) {
  long long int d_sec =  finish->tv_sec - start->tv_sec; 
  long long int d_nsec =  finish->tv_nsec - start->tv_nsec; 

  if(d_nsec < 0 ) {
    d_sec--;
    d_nsec += 1000000000; 
  } 
  *difference = d_sec * 1000000000 + d_nsec;
  return !(*difference > 0);
}


int main() {

	struct  timespec start, finish;
	long long int time_elapsed;
	clock_gettime(CLOCK_MONOTONIC, &start);

	kernel <<<26,26>>>();
	hipDeviceSynchronize();

	clock_gettime(CLOCK_MONOTONIC, &finish);
	time_difference(&start, &finish, &time_elapsed);
	printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9)); 

	return 0;
}


